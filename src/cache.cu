/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#define SIZE (2048*1024/sizeof(int))

__device__ int *nonce;

// called by host, executed by GPU
__global__ void init() {
	nonce = (int *)malloc(SIZE*sizeof(int));
}

__global__ void setVals() {
	hiprandState_t state;

	/* we have to initialize the state */
	hiprand_init(0, /* the seed controls the sequence of random values that are produced */
			  0, /* the sequence number is only important with multiple cores */
			  0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
			  &state);
	for(int i=0;i<SIZE;i++){
		int r = hiprand(&state) % SIZE;
		//printf("%d ", r);
		*(nonce+r) = i;
	}
}

__global__ void getVals() {
	int j;
	for(int i=0;i<SIZE;i++){
		j = *(nonce+i);
		//printf("%d ", j);
	}
}

int main(void) {

	//printf("%d\n", sizeof(int));
	init<<<1, 1>>>();
	getVals<<<1, 1>>>();
	setVals<<<1, 1>>>();

	return 0;
}
